#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include "helper_functions.h"
#include <fstream>
#include <iostream>

__global__ void add (int *a,int *b, int *c,int N) 
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < N) 
	{
		c[tid] = a[tid]+b[tid];
	}
}

void add_host(int *a,int *b,int *d,int N)
{
	int i;
	for (i = 0; i < N ; i ++ ) 
	{
		d[i] = a[i] + b[i];	
	}	
}

bool is_same(int *c,int *d,int N)
{
	bool same_tab = true;
	for (int i=0; i< N; i++ )
	{
		same_tab &= c[i] == d[i];
	}
	return same_tab;
}

void usage(void) {
	printf("Usage:\n./a.out size thread_per_block block_per_grid\n");
	exit(0);
}

int main(int argc,char **argv)
{
	if (argc != 4) {
		usage();
	}
	int *a,*b,*c,*d;
	int N = atoi(argv[1]);	
	int thread_per_block = atoi(argv[2]);
	int block_per_grid = atoi(argv[3]);
	a = (int*)malloc(N*sizeof(int));
	b = (int*)malloc(N*sizeof(int));
	c = (int*)malloc(N*sizeof(int));
	d = (int*)malloc(N*sizeof(int));

	for (int i=0;i<N;i++) 
	{
		a[i] = i;
		b[i] = i*1;
	}

	std::ofstream cpuFile;
	std::ofstream gpuFile;

	cpuFile.open("cpu.txt");
	gpuFile.open("gpu.txt");

	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);

	int *dev_a, *dev_b, *dev_c;
	hipMalloc((void**)&dev_a,N * sizeof(int));
	hipMalloc((void**)&dev_b,N * sizeof(int));
	hipMalloc((void**)&dev_c,N * sizeof(int));
	hipMemcpy(dev_a, a , N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b , N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c , N*sizeof(int),hipMemcpyHostToDevice);
	add<<<block_per_grid,thread_per_block>>>(dev_a,dev_b,dev_c,N);
	hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);

	sdkStopTimer(&timer);
	float time = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

// Host part

	StopWatchInterface *timer_host = NULL;
	sdkCreateTimer(&timer_host);
	sdkResetTimer(&timer_host);
	sdkStartTimer(&timer_host);

	add_host(a,b,d,N);

	sdkStopTimer(&timer_host);
	float time_host = sdkGetTimerValue(&timer_host);
	sdkDeleteTimer(&timer_host);

// Checking if same
	bool same_host_gpu = is_same(c,d,N);
	if (same_host_gpu){ 
		printf("GPU: %d %f\n",N,time);	
		printf("CPU: %d %f\n",N,time_host);	

		gpuFile << N << " " << time << std::endl;
		cpuFile << N << " " << time_host << std::endl;
	}
	else
	{
		printf("SIZE:%d ERROR\n", N);
	}
	free(a);
	free(b);
	free(c);
	free(d);
// Happy end
	cpuFile.close();
	gpuFile.close();

	std::cin.get();
	std::cin.ignore();
	return 0;
}
