#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include "helper_functions.h"

__global__ void add (int *a,int *b, int *c,int N) 
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < N) 
	{
		c[tid] = a[tid]+b[tid];
	}
}

void add_host(int *a,int *b,int *d,int N)
{
	int i;
	for (i = 0; i < N ; i ++ ) 
	{
		d[i] = a[i] + b[i];	
	}	
}

bool is_same(int *c,int *d,int N)
{
	bool same_tab = true;
	for (int i=0; i< N; i++ )
	{
		same_tab &= c[i] == d[i];
	}
	return same_tab;
}

void usage(void) {
	printf("Usage:\n./a.out size thread_per_block block_per_grid\n");
	exit(0);
}

int main(int argc,char **argv)
{
	if (argc != 4) {
		usage();
	}
	int *a,*b,*c,*d;
	int N = atoi(argv[1]);	
	int thread_per_block = atoi(argv[2]);
	int block_per_grid = atoi(argv[3]);
	a = (int*)malloc(N*sizeof(int));
	b = (int*)malloc(N*sizeof(int));
	c = (int*)malloc(N*sizeof(int));
	d = (int*)malloc(N*sizeof(int));

	for (int i=0;i<N;i++) 
	{
		a[i] = i;
		b[i] = i*1;
	}

	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);

	int *dev_a, *dev_b, *dev_c;
	hipMalloc((void**)&dev_a,N * sizeof(int));
	hipMalloc((void**)&dev_b,N * sizeof(int));
	hipMalloc((void**)&dev_c,N * sizeof(int));
	hipMemcpy(dev_a, a , N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b , N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c , N*sizeof(int),hipMemcpyHostToDevice);
	add<<<block_per_grid,thread_per_block>>>(dev_a,dev_b,dev_c,N);
	hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);

	sdkStopTimer(&timer);
	float time = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);
	
	printf("----------Result for kernel----------\n");
	for (int i=0;i<N;i++) 
	{
		printf("%d+%d=%d\n",a[i],b[i],c[i]);
	}
	printf("Time for the kernel: %f ms\n",time);	

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

// Host part

	add_host(a,b,d,N);
	printf("----------Result for host:-----------\n");
	for (int i=0;i<N;i++)
	{
		printf("%d+%d=%d\n",a[i],b[i],d[i]);
	}

// Checking if same
	bool same_host_gpu = is_same(c,d,N);
	if (same_host_gpu) 
		printf("Same result for host and kernel\n");
	else
		printf("Different result for host and kernel\n");	

// Happy end
	return 0;
}
