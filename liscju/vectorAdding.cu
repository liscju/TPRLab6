#include <cstdio>
#include <hip/hip_runtime.h>
#include <cstdlib>

__global__ void add (int *a,int *b, int *c,int N) 
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < N) 
	{
		c[tid] = a[tid]+b[tid];
	}
}

void usage(void) {
	printf("Usage:\n./a.out size\n");
	exit(0);
}

int main(int argc,char **argv)
{
	if (argc < 2) {
		usage();
	}
	int *a,*b,*c;
	int N = atoi(argv[1]);	
	a = (int*)malloc(N*sizeof(int));
	b = (int*)malloc(N*sizeof(int));
	c = (int*)malloc(N*sizeof(int));

	int *dev_a, *dev_b, *dev_c;
	hipMalloc((void**)&dev_a,N * sizeof(int));
	hipMalloc((void**)&dev_b,N * sizeof(int));
	hipMalloc((void**)&dev_c,N * sizeof(int));
	for (int i=0;i<N;i++) 
	{
		a[i] = i;
		b[i] = i*1;
	}
	hipMemcpy(dev_a, a , N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b , N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c , N*sizeof(int),hipMemcpyHostToDevice);
	add<<<1,N>>>(dev_a,dev_b,dev_c,N);
	hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);
	for (int i=0;i<N;i++) 
	{
		printf("%d+%d=%d\n",a[i],b[i],c[i]);
	}
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}
