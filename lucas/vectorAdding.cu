#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include "helper_functions.h"

#define MY_CPU 0
#define MY_GPU 1

// GLOBALS
int type;
int vectorSize;
int blockCount;
int threadCount;
	
int* a;
int* b;
int* c;
// END GLOBALS

void init(int *a, int *b, int size) {
	int i;
	for (i=0; i<size; i++) {
		a[i] = i;
		b[i] = i;
	}
}

__global__ void add(int *a,int *b, int *c, int size) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < size) {
		c[tid] = a[tid] + b[tid];
	}
}

void addCPU(int *a,int *b, int *c, int size) {
	int i;
	for(i=0; i<size; i++) {
		c[i] = a[i]+b[i];
	}
}

void countCPU() {
	printf("###CPU:\n %d %d", vectorSize, blockCount);
	StopWatchInterface* timer = NULL;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);
	
	addCPU(a, b, c, vectorSize);
	
	sdkStopTimer(&timer);
	float time = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);
	
	printf("%d+%d=%d\n", a[vectorSize-1], b[vectorSize-1], c[vectorSize-1]);
	printf("%d %f\n", vectorSize, time);
}

void countGPU() {
	printf("###GPU:\n %d %d", vectorSize, blockCount);
	int *dev_a, *dev_b, *dev_c;
	hipMalloc((void**)&dev_a, vectorSize * sizeof(int));
	hipMalloc((void**)&dev_b, vectorSize * sizeof(int));
	hipMalloc((void**)&dev_c, vectorSize * sizeof(int));
	
	hipMemcpy(dev_a, a, vectorSize * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, vectorSize * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, vectorSize * sizeof(int), hipMemcpyHostToDevice);
	
	StopWatchInterface* timer = NULL;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);

	add<<<blockCount,threadCount>>>(dev_a, dev_b, dev_c, vectorSize);
	hipDeviceSynchronize();
	
	sdkStopTimer(&timer);
	float time = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);

	hipMemcpy(c, dev_c, vectorSize * sizeof(int), hipMemcpyDeviceToHost);
	
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	printf("%d+%d=%d\n", a[vectorSize-1], b[vectorSize-1], c[vectorSize-1]);
	printf("%d %f\n", vectorSize, time);
}

int main(int argc, char** argv) {
	if(argc != 4) {
		fprintf(stderr, "Wrong arguments. Usage: %s <vector-size> <block-count> <thread-count>\n", argv[0]);
		return EXIT_FAILURE;
	}
	int vectorSize = atoi(argv[1]);
	int blockCount = atoi(argv[2]);
	int threadCount = atoi(argv[3]);
	
	int* a;
	int* b;
	int* c;

	a = (int*)malloc(vectorSize * sizeof(int));
	b = (int*)malloc(vectorSize * sizeof(int));
	c = (int*)malloc(vectorSize * sizeof(int));

	init(a, b, vectorSize);
	
	countGPU();
	countCPU();

	free(a);
	free(b);
	free(c);	

	return EXIT_SUCCESS;
}


