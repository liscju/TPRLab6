#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <cstdio>
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"
#include <fstream>


void doStuffOnCPU() //does stuff on CPU
{

}


__global__ void add (int *a,int *b, int *c, const int N) 
{
	long long int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < N) 
	{
		c[tid] = a[tid]+b[tid];
	}
}

int main(int argc, char** argv)
{
	const int dimension = 6;
	const int threads[dimension] = { 32, 64, 128, 256, 512, 1024 };
	const int blocks[dimension] = { 128, 256, 512, 1024, 2048, 4096 };
	const int sizes[dimension] = { 10000, 100000, 1000000, 10000000, 100000000, 1000000000 };

	StopWatchInterface *timer = NULL;
	float elapsedTime = 0.0f;
	
	std::ofstream resultsFile;
	resultsFile.open("results.txt");
	char * tempTableSize = argv[1];
	unsigned int tableSize = *tempTableSize - '0'; //that shit is dirty


	int N = tableSize;
	int * a;
	int * b;
	int * c;

	int threadId = 0;
	int block = 0;
	int *dev_a, *dev_b, *dev_c;
	hipEvent_t start, stop;

	for (int sizeLoop = 0; sizeLoop < dimension; sizeLoop++)
	{
		a = new int[sizes[sizeLoop]];
		b = new int[sizes[sizeLoop]];
		c = new int[sizes[sizeLoop]];
		for (int i = 0; i < dimension; i++)
		{
			threadId = threads[i];
			for (int x = 0; x < dimension; x++)
			{
				block = blocks[x];

				hipMalloc((void**)&dev_a, N * sizeof(int));
				hipMalloc((void**)&dev_b, N * sizeof(int));
				hipMalloc((void**)&dev_c, N * sizeof(int));
				for (int j = 0; j < N; j++)
				{
					a[j] = j;
					b[j] = j * 1;
				}

				sdkCreateTimer(&timer);
				checkCudaErrors(hipEventCreate(&start));
				checkCudaErrors(hipEventCreate(&stop));

				checkCudaErrors(hipEventRecord(start, 0));
				hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
				hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
				hipMemcpy(dev_c, c, N*sizeof(int), hipMemcpyHostToDevice);

				add << <block, threadId >> >(dev_a, dev_b, dev_c, N);

				hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
				checkCudaErrors(hipEventRecord(stop, 0));
				checkCudaErrors(hipDeviceSynchronize());
				sdkStopTimer(&timer);
				checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));

				resultsFile << "Time: " << elapsedTime << "\tBlocks: " << block << "\tThreads: " << threadId << "\tProblem size: " << sizes[sizeLoop] << std::endl;
				hipFree(dev_a);
				hipFree(dev_b);
				hipFree(dev_c);
			}
		}
		delete[] a;
		delete[] b;
		delete[] c;
	}
	resultsFile.close();
	std::cin.get();
	std::cin.ignore();
	return 0;
}
