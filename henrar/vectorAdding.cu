#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <cstdio>
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"
#include <fstream>



void doStuffOnCPU() //does stuff on CPU
{

}


__global__ void add (int *a,int *b, int *c, const int N) 
{
	long long int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < N) 
	{
		c[tid] = a[tid]+b[tid];
	}
}

int main(int argc, char** argv)
{
	if (argc != 2) {
		fprintf(stderr, "Wrong arguments. \n", argv[0]);
		std::cin.get();
		std::cin.ignore();
		return EXIT_FAILURE;
	}
	StopWatchInterface *timer = NULL;
	float elapsedTime = 0.0f;
	
	
	int threadId = 1024;
	int block = 65535;

	char * tempTableSize = argv[1];
	unsigned int tableSize = *tempTableSize - '0'; //that shit is dirty
	
	int N = tableSize;

	int * a = new int[N];
	int * b = new int[N];
	int * c = new int[N];

	int *dev_a, *dev_b, *dev_c;
	hipMalloc((void**)&dev_a,N * sizeof(int));
	hipMalloc((void**)&dev_b,N * sizeof(int));
	hipMalloc((void**)&dev_c,N * sizeof(int));
	for (int i=0;i<N;i++) 
	{
		a[i] = i;
		b[i] = i*1;
	}

	hipEvent_t start, stop;
	sdkCreateTimer(&timer);
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	checkCudaErrors(hipEventRecord(start, 0));
	hipMemcpy(dev_a, a , N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b , N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c , N*sizeof(int),hipMemcpyHostToDevice);
	
	add<<<block,threadId>>>(dev_a, dev_b, dev_c, N);
	
	hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipDeviceSynchronize());
	sdkStopTimer(&timer);
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));


	//print stuff in format:
	//"%d %f", size, time
	/*for (int i=0;i<N;i++) 
	{
		printf("%d+%d=%d\n",a[i],b[i],c[i]);
	}*/

	std::cout << "Program finished in time: " << elapsedTime << std::endl;
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	std::cin.get();
	std::cin.ignore();
	return 0;
}
