#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <cstdio>
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

__global__ void add (int *a,int *b, int *c, const int N) 
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < N) 
	{
		c[tid] = a[tid]+b[tid];
	}
}

int main(void)
{
	StopWatchInterface *timer = NULL;
	float elapsedTime = 0.0f;

	int tableSize = 0;
	std::cout << "Podaj rozmiar tablicy: ";
	std::cin >> tableSize;
	std::cout << std::endl;	
	
	int N = tableSize;

	int a[N],b[N],c[N];
	int *dev_a, *dev_b, *dev_c;
	hipMalloc((void**)&dev_a,N * sizeof(int));
	hipMalloc((void**)&dev_b,N * sizeof(int));
	hipMalloc((void**)&dev_c,N * sizeof(int));
	for (int i=0;i<N;i++) 
	{
		a[i] = i;
		b[i] = i*1;
	}

	hipEvent_t start, stop;
	sdkCreateTimer(&timer);
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	checkCudaErrors(hipEventRecord(start, 0));
	hipMemcpy(dev_a, a , N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b , N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c , N*sizeof(int),hipMemcpyHostToDevice);
	add<<<1,N>>>(dev_a,dev_b,dev_c, N);
	hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipDeviceSynchronize());
	sdkStopTimer(&timer);
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
	for (int i=0;i<N;i++) 
	{
		printf("%d+%d=%d\n",a[i],b[i],c[i]);
	}
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	std::cout << "Elapsed time: " << elapsedTime << std::endl;
	return 0;
}
